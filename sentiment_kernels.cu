#include "hip/hip_runtime.h"
#include "sentiment_kernels.h"

#include <ctime>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "cuda_primitives.h"

/************************/
/***** CUDA Kernels *****/
/************************/

__global__ void
setupRandomVectorGen(hiprandState* state, unsigned long seed, unsigned int numElems) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= numElems) return;
    hiprand_init(seed, id, 0, &(state[id]));
}

__global__ void
runRandomVectorGen(float* vec, hiprandState* globalState, float threshold, unsigned int numElems) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= numElems) return;
    hiprandState localState = globalState[id];
    float rndVal = hiprand_uniform(&localState);
    // scale to (-threshold, +threshold)
    vec[id] = (rndVal * 2 * threshold) - threshold;
}

__global__ void
updateParams(float* params, float* derivatives, float* weights, float learningRate, unsigned int numElems) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= numElems) return;
    float epsilon = 0.0001;
    weights[id] += derivatives[id]*derivatives[id];
    params[id] -= (learningRate * derivatives[id])/(sqrt(weights[id]) + epsilon);
}

/***************************/
/***** Kernel Wrappers *****/
/***************************/

void
kernelRandomWordVectors(cParamMem_t& params, float threshold) {
    timeval tim;
    gettimeofday(&tim, NULL);
    double t1=tim.tv_sec+(tim.tv_usec/1000000.0);
    
    // hipEvent_t sync_event;
    // checkCudaErrors(hipEventCreate(&sync_event));

    unsigned int blockSize = 1024;
    unsigned int numElems = params.numWords * params.wordDim;
    unsigned int numBlocks = numElems / blockSize + 1;
    dim3 threadsPerBlock(blockSize, 1, 1);
    hiprandState* devState;
    checkCudaErrors(hipMalloc((void**)&devState, numElems*sizeof(hiprandState)));
    setupRandomVectorGen<<<numBlocks, threadsPerBlock>>>(devState, time(NULL), numElems);
    checkCudaErrors(hipGetLastError());
    runRandomVectorGen<<<numBlocks, threadsPerBlock>>>(params.wordVectors, devState, threshold, numElems);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(devState));

    // checkCudaErrors(hipEventRecord(sync_event));
    // checkCudaErrors(hipEventSynchronize(sync_event));

    gettimeofday(&tim, NULL);
    double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
    printf("Random word vectors time: %f\n", t2-t1);
}

void
kernelUpdateParams(cParamMem_t& params, cParamMem_t& derivatives,
  cParamMem_t& adagradWts, float learningRate) {
    timeval tim;
    gettimeofday(&tim, NULL);
    double t1=tim.tv_sec+(tim.tv_usec/1000000.0);

    // hipEvent_t sync_event;
    // checkCudaErrors(hipEventCreate(&sync_event));

    unsigned int blockSize = 1024;
    unsigned int numElems = params.totalSize;
    unsigned int numBlocks = numElems / blockSize + 1;
    dim3 threadsPerBlock(blockSize, 1, 1);
    updateParams<<<numBlocks,threadsPerBlock>>>(params.base, derivatives.base,
        adagradWts.base, learningRate, params.totalSize);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    // checkCudaErrors(hipEventRecord(sync_event));
    // checkCudaErrors(hipEventSynchronize(sync_event));

    gettimeofday(&tim, NULL);
    double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
    printf("Update params time: %f\n", t2-t1);
}
